
#include "cuda_types.h"
#include "cuda_utils.h"
#include "sparse_dataset.h"

void initMatDescriptors( DeviceDataset *d )
{
	//Train
	cusparseCheckError ( hipsparseCreateMatDescr(&(d->spTrain.descr)) ); 
	cusparseCheckError ( hipsparseSetMatIndexBase(d->spTrain.descr, HIPSPARSE_INDEX_BASE_ZERO) );
	cusparseCheckError ( hipsparseSetMatType(d->spTrain.descr, HIPSPARSE_MATRIX_TYPE_GENERAL) );

	//Test
	cusparseCheckError ( hipsparseCreateMatDescr(&(d->spTest.descr)) ); 
	cusparseCheckError ( hipsparseSetMatIndexBase(d->spTest.descr, HIPSPARSE_INDEX_BASE_ZERO) );
	cusparseCheckError ( hipsparseSetMatType(d->spTest.descr, HIPSPARSE_MATRIX_TYPE_GENERAL) );
}

void initMatDescriptorsForSampling( DeviceDataset *d ) {

	//SubSampling - Hessian
	cusparseCheckError ( hipsparseCreateMatDescr(&(d->spHessianSample.descr)) ); 
	cusparseCheckError ( hipsparseSetMatIndexBase(d->spHessianSample.descr, HIPSPARSE_INDEX_BASE_ZERO) );
	cusparseCheckError ( hipsparseSetMatType(d->spHessianSample.descr, HIPSPARSE_MATRIX_TYPE_GENERAL) );

	//gradient
	cusparseCheckError ( hipsparseCreateMatDescr(&(d->spGradientSample.descr)) ); 
	cusparseCheckError ( hipsparseSetMatIndexBase(d->spGradientSample.descr, HIPSPARSE_INDEX_BASE_ZERO) );
	cusparseCheckError ( hipsparseSetMatType(d->spGradientSample.descr, HIPSPARSE_MATRIX_TYPE_GENERAL) );
}

void initMatDescriptorsForSparseSampling( DeviceDataset *d ) {

	//SubSampling - Hessian
	cusparseCheckError ( hipsparseCreateMatDescr(&(d->spSampledHessianTrain.descr)) ); 
	cusparseCheckError ( hipsparseSetMatIndexBase(d->spSampledHessianTrain.descr, HIPSPARSE_INDEX_BASE_ZERO) );
	cusparseCheckError ( hipsparseSetMatType(d->spSampledHessianTrain.descr, HIPSPARSE_MATRIX_TYPE_GENERAL) );

	//gradient
	cusparseCheckError ( hipsparseCreateMatDescr(&(d->spSampledGradientTrain.descr)) ); 
	cusparseCheckError ( hipsparseSetMatIndexBase(d->spSampledGradientTrain.descr, HIPSPARSE_INDEX_BASE_ZERO) );
	cusparseCheckError ( hipsparseSetMatType(d->spSampledGradientTrain.descr, HIPSPARSE_MATRIX_TYPE_GENERAL) );
}

void convertGradientSampleToCSR (SparseDataset *spGradientSample, int sampleSize, int cols, real *devPtr) {

	//make sure that the data is sorted here. 
	size_t pBufferSizeInBytes = 0; 
	void* pBuffer = (void *)devPtr; 

	//Sampled Dataset Here. 
	cusparseCheckError( 
			hipsparseXcoosort_bufferSizeExt( 
				cusparseHandle, sampleSize, cols, spGradientSample->nnz, 
				spGradientSample->rowPtr, spGradientSample->colPtr, &pBufferSizeInBytes ) ); 

	cusparseCheckError( 
		hipsparseCreateIdentityPermutation( cusparseHandle, spGradientSample->nnz, spGradientSample->P) ); 
	
	cusparseCheckError( 
		hipsparseXcoosortByRow( cusparseHandle, sampleSize, cols, spGradientSample->nnz, 
				spGradientSample->rowPtr, spGradientSample->colPtr, spGradientSample->P, pBuffer ) ); 

	cusparseCheckError( 
		hipsparseDgthr( cusparseHandle, spGradientSample->nnz, spGradientSample->valPtr, 
				spGradientSample->sortedVals, spGradientSample->P, HIPSPARSE_INDEX_BASE_ZERO ) ); 

	//convert to csr format. 
	cusparseCheckError( 
			hipsparseXcoo2csr( cusparseHandle, spGradientSample->rowPtr, spGradientSample->nnz, sampleSize, 
				spGradientSample->rowCsrPtr, HIPSPARSE_INDEX_BASE_ZERO ) 
		); 	

	//fprintf( stderr, "Converting gradient to CSR .... \n"); 
}


void convertHessianSampleToCSR (SparseDataset *spHessianSample, int sampleSize, int cols, real *devPtr) {

	//make sure that the data is sorted here. 
	size_t pBufferSizeInBytes = 0; 
	void* pBuffer = (void *)devPtr; 

	//Sampled Dataset Here. 
	cusparseCheckError( 
			hipsparseXcoosort_bufferSizeExt( 
				cusparseHandle, sampleSize, cols, spHessianSample->nnz, 
				spHessianSample->rowPtr, spHessianSample->colPtr, &pBufferSizeInBytes ) ); 

	cusparseCheckError( 
		hipsparseCreateIdentityPermutation( cusparseHandle, spHessianSample->nnz, spHessianSample->P) ); 
	
	cusparseCheckError( 
		hipsparseXcoosortByRow( cusparseHandle, sampleSize, cols, spHessianSample->nnz, 
				spHessianSample->rowPtr, spHessianSample->colPtr, spHessianSample->P, pBuffer ) ); 

	cusparseCheckError( 
		hipsparseDgthr( cusparseHandle, spHessianSample->nnz, spHessianSample->valPtr, 
				spHessianSample->sortedVals, spHessianSample->P, HIPSPARSE_INDEX_BASE_ZERO ) ); 

	//convert to csr format. 
	cusparseCheckError( 
			hipsparseXcoo2csr( cusparseHandle, spHessianSample->rowPtr, spHessianSample->nnz, sampleSize, 
				spHessianSample->rowCsrPtr, HIPSPARSE_INDEX_BASE_ZERO ) 
		); 	

	//fprintf( stderr, "Converting hessian to CSR .... \n"); 
}

void convertToCSR( DeviceDataset *d, real *devPtr )
{
	//make sure that the data is sorted here. 
	size_t pBufferSizeInBytes = 0; 
	void* pBuffer = (void *)devPtr; 

	//Train Dataset Here. 
	cusparseCheckError( 
			hipsparseXcoosort_bufferSizeExt( 
				cusparseHandle, d->rows, d->cols, d->spTrain.nnz, 
				d->spTrain.rowPtr, d->spTrain.colPtr, &pBufferSizeInBytes ) ); 
	fprintf( stderr, "Memory needed to sort coo data --> %d \n", pBufferSizeInBytes ); 

	cusparseCheckError( 
		hipsparseCreateIdentityPermutation( cusparseHandle, d->spTrain.nnz, d->spTrain.P) ); 
	
	cusparseCheckError( 
		hipsparseXcoosortByRow( cusparseHandle, d->rows, d->cols, d->spTrain.nnz, 
				d->spTrain.rowPtr, d->spTrain.colPtr, d->spTrain.P, pBuffer ) ); 

	cusparseCheckError( 
		hipsparseDgthr( cusparseHandle, d->spTrain.nnz, d->spTrain.valPtr, 
				d->spTrain.sortedVals, d->spTrain.P, HIPSPARSE_INDEX_BASE_ZERO ) ); 

	//convert to csr format. 
	cusparseCheckError( 
			hipsparseXcoo2csr( cusparseHandle, d->spTrain.rowPtr, d->spTrain.nnz, d->rows, 
				d->spTrain.rowCsrPtr, HIPSPARSE_INDEX_BASE_ZERO ) 
		); 	


	//Test Dataset here. 
	cusparseCheckError( 
			hipsparseXcoosort_bufferSizeExt( 
				cusparseHandle, d->rows, d->cols, d->spTest.nnz, 
				d->spTest.rowPtr, d->spTest.colPtr, &pBufferSizeInBytes ) ); 
	fprintf( stderr, "Memory needed to sort coo data --> %d \n", pBufferSizeInBytes ); 

	cusparseCheckError( 
		hipsparseCreateIdentityPermutation( cusparseHandle, d->spTest.nnz, d->spTest.P) ); 
	
	cusparseCheckError( 
		hipsparseXcoosortByRow( cusparseHandle, d->rows, d->cols, d->spTest.nnz, 
				d->spTest.rowPtr, d->spTest.colPtr, d->spTest.P, pBuffer ) ); 

	cusparseCheckError( 
		hipsparseDgthr( cusparseHandle, d->spTest.nnz, d->spTest.valPtr, 
				d->spTest.sortedVals, d->spTest.P, HIPSPARSE_INDEX_BASE_ZERO ) ); 

	//convert to csr format. 
	cusparseCheckError( 
			hipsparseXcoo2csr( cusparseHandle, d->spTest.rowPtr, d->spTest.nnz, d->rows, 
				d->spTest.rowCsrPtr, HIPSPARSE_INDEX_BASE_ZERO ) 
		); 	

/*
	cusparseCheckError( 
			hipsparseXcoo2csr( cusparseHandle, d->spTest.rowPtr, d->spTest.nnz, d->testSize, 
				d->spTest.rowCsrPtr, HIPSPARSE_INDEX_BASE_ZERO ) 
		); 	

	//convert the csr matrix to csc matrix here. 
	cusparseCheckError( 
			hipsparseDcsr2csc( cusparseHandle, d->rows, d->cols, d->spTrain.nnz, 
					d->spTrain.valPtr, d->spTrain.rowCsrPtr, d->spTrain.colPtr, 
					d->spTrain.cscValPtr, d->spTrain.cscRowPtr, d->spTrain.cscColPtr, 
					HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO ) ); 
*/
}
